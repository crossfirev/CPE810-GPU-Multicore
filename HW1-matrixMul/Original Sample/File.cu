// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
using namespace std;

int main()
{
	// Array size of 2^16 (65536 elements)
	int N = 256;
	size_t bytes = sizeof(int) * N;

	// Vectors for holding the host-side (CPU-side) data
	vector<int> a;		a.reserve(N);
	vector<int> b;		b.reserve(N);
	vector<int> c;		c.reserve(N);

	// Initialize random numbers in each array
	for (int i = 0; i < N; i++)
	{
		a.push_back(rand() % 100);
		b.push_back(rand() % 100);
	}

	// Allocate memory on the device
	int* d_a, * d_b, * d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Copy data from the host to the device (CPU -> GPU)
	hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

	// Threads per CTA (1024)
	int NUM_THREADS = 1 << 10;

	// CTAs per Grid
	// We need to launch at LEAST as many threads as we have elements
	// This equation pads an extra CTA to the grid if N cannot evenly be divided
	// by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	// Launch the kernel on the GPU
	// Kernel calls are asynchronous (the CPU program continues execution after
	// call, but no necessarily before the kernel finishes)
	vectorAdd <<<NUM_BLOCKS, NUM_THREADS >>> (d_a, d_b, d_c, N);

	// Copy sum vector from device to host
	// hipMemcpy is a synchronous operation, and waits for the prior kernel
	// launch to complete (both go to the default stream in this case).
	// Therefore, this hipMemcpy acts as both a memcpy and synchronization
	// barrier.
	hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

	// Check result for errors
	verify_result(a, b, c);

	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
}